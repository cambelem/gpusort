#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <openssl/md5.h>
#include "d_hybrid_sort.h"
#include "CHECK.h"
#include "config.h"
#include "wrappers.h"

//prototype for the kernel
__global__ void d_sort_kernel();

/*d_crack
*
* Sets up and calls the kernal to brute-force a password hash.
*
* @params
*   hash    - the password hash to brute-force
*   hashLen - the length of the hash
*   outpass - the result password to return
*/
float d_sort() {

    hipEvent_t start_cpu, stop_cpu;
    float cpuMsecTime = -1;

    //Use cuda functions to do the timing
    //create event objects
    CHECK(hipEventCreate(&start_cpu));
    CHECK(hipEventCreate(&stop_cpu));
    //record the starting time
    CHECK(hipEventRecord(start_cpu));

    // unsigned long size = 2 * NUMCHARS * sizeof(unsigned char);
    // unsigned long outsize = pow(NUMCHARS, 2) * 3;

    // unsigned char * d_passwords;
    // CHECK(hipMalloc((void**)&d_passwords, size));
    // unsigned char * d_result;
    // CHECK(hipMalloc((void**)&d_result, outsize));


    //Copy the starting passwords array and valid characters to the GPU
    // CHECK(hipMemcpyToSymbol(HIP_SYMBOL(VALID_CHARS), VALID_CHARS_CPU, NUMCHARS * sizeof(char)));
    // CHECK(hipMemcpy(d_passwords, STARTING_PASSES, 2 * NUMCHARS, hipMemcpyHostToDevice));


    // Beginning of Four-way Radix Sort
    // We need a block size of 256
    // dim3 block(BLOCKDIM, 1, 1);
    // dim3 grid(1, 1, 1);

    // d_generate_kernel<<<grid, block>>>(d_passwords, 1, NUMCHARS, d_result);

    // CHECK(hipDeviceSynchronize());
    //
    // unsigned char * passwords = (unsigned char *) Malloc(outsize);
    // CHECK(hipMemcpy(passwords, d_result, outsize, hipMemcpyDeviceToHost));
    //
    // CHECK(hipFree(d_passwords));
    // CHECK(hipFree(d_result));
    //
    // free(passwords);

    //record the ending time and wait for event to complete
    CHECK(hipEventRecord(stop_cpu));
    CHECK(hipEventSynchronize(stop_cpu));
    //calculate the elapsed time between the two events
    CHECK(hipEventElapsedTime(&cpuMsecTime, start_cpu, stop_cpu));
    return cpuMsecTime;
}

/*d_generate_kernel
*  Kernal code executed by each thread to generate a list of all possible
*  passwords of length n + 1.  To do this, each thread will work on one element
*  in passwords and append all characters in VALID_CHARS to it. This kernal
*  works in place, so it will alter the input array.
*
*  @params:
*   passwords - array filled with current passwords to build off of.
*   length    - length of the given passwords
*   n         - number of items currently in passwords array
*   d_result  - location to place newly generated passwords.
*/
__global__ void d_sort_kernel() {
  // unsigned long index = blockIdx.x * blockDim.x + threadIdx.x;
}
