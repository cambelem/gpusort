#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <openssl/md5.h>
#include "d_hybrid_sort.h"
#include "CHECK.h"
#include "config.h"
#include "wrappers.h"

#define NUMBER_OF_PROCESSORS 1024
#define BLOCK_DIM 256

//prototype for pivot counting kernal
__global__ void d_count_kernel(unsigned int * d_pivots,
  int * r_buckets, int pivotsLength, unsigned int * r_indices,
  unsigned int * r_sublist, unsigned int * d_in, int itemCount);

//prototype for bucketsort
__global__ void d_bucketsort(unsigned int * d_in, unsigned int * d_indices,
    unsigned int * d_sublist, unsigned int * r_outputlist,
    unsigned int * d_bucketoffsets, int itemCount);

//prototype for the kernel
__global__ void d_sort_kernel();

/*d_crack
*
* Sets up and calls the kernal to brute-force a password hash.
*
* @params
*   hash    - the password hash to brute-force
*   hashLen - the length of the hash
*   outpass - the result password to return
*/
float d_sort(unsigned int * in, unsigned int length) {

    hipEvent_t start_cpu, stop_cpu;
    float cpuMsecTime = -1;

    //Use cuda functions to do the timing
    //create event objects
    CHECK(hipEventCreate(&start_cpu));
    CHECK(hipEventCreate(&stop_cpu));
    //record the starting time
    CHECK(hipEventRecord(start_cpu));

    //Find min and max
    unsigned int max = 0;
    unsigned int min = UINT_MAX;
    for (unsigned int i = 0; i < length; i++) {
      if (in[i] < min) {
        min = in[i];
      }
      if (in[i] > max) {
        max = in[i];
      }
    }

    //Compute pivots through linear interpolation
    unsigned int pivotsLength = (NUMBER_OF_PROCESSORS * 2) - 1;
    unsigned int * pivots = new unsigned int[pivotsLength];
    int * buckets_count = new int[pivotsLength];
    int slope = (max - min)/pivotsLength;
    unsigned int j = 0;
    for (unsigned int i = 0; i < pivotsLength; i++) {
      pivots[i] = (slope * j);
      buckets_count[i] = 0;
      j += length/pivotsLength;
    }

    /****************************STEP 1****************************************/
    //Launch a kernal to count the number of items in each bucket so we can
    //refine our pivots later.

    //Input/output mallocs
    unsigned int * d_pivots;
    CHECK(hipMalloc((void**)&d_pivots, pivotsLength * sizeof(unsigned int)));
    int * r_buckets;
    CHECK(hipMalloc((void**)&r_buckets, pivotsLength * sizeof(int)));
    unsigned int * d_in;
    CHECK(hipMalloc((void**)&d_in, length * sizeof(unsigned int)));
    unsigned int * r_indices;
    CHECK(hipMalloc((void**)&r_indices, length * sizeof(unsigned int)));
    unsigned int * r_sublist;
    CHECK(hipMalloc((void**)&r_sublist,
      (pivotsLength + 1) * sizeof(unsigned int)));

    //Copying things to memory
    CHECK(hipMemcpy(d_pivots, pivots,
      pivotsLength * sizeof(unsigned int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_in, in,
      length * sizeof(unsigned int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(r_buckets, buckets_count,
      pivotsLength * sizeof(int), hipMemcpyHostToDevice));

    //kernel dimensions
    dim3 block(BLOCKDIM, 1, 1);
    dim3 grid(ceil((float) length/BLOCKDIM), 1, 1);

    //Launching kernel
    d_count_kernel<<<grid, block>>>(d_pivots, r_buckets, pivotsLength,
      r_indices, r_sublist, d_in, length);

    CHECK(hipDeviceSynchronize());

    int * buckets = (int *) Malloc(pivotsLength * sizeof(int));
    CHECK(hipMemcpy(buckets, r_buckets, pivotsLength * sizeof(unsigned int), hipMemcpyDeviceToHost));
    unsigned int * indices = (unsigned int *) Malloc(length * sizeof(unsigned int));
    CHECK(hipMemcpy(indices, r_indices, length * sizeof(unsigned int), hipMemcpyDeviceToHost));
    unsigned int * sublist = (unsigned int *) Malloc(length * sizeof(unsigned int));
    hipMemcpy(sublist, r_sublist, length * sizeof(unsigned int), hipMemcpyDeviceToHost);

    CHECK(hipFree(d_pivots));
    CHECK(hipFree(r_buckets));
    CHECK(hipFree(r_indices));
    CHECK(hipFree(r_sublist));
    CHECK(hipFree(d_in));

    /***************************STEP 1 COMPLETE********************************/

    // int count = 0;
    // for (int i = 0; i < length; i++) {
    //   if (in[i] == 2034) {
    //     count++;
    //   }
    // }
    // std::cout << count << std::endl;

    // for (unsigned int i = 0; i < pivotsLength; i++) {
    //   std::cout << buckets[i] << std::endl;
    // }

    /***************************STEP 2*****************************************/
    // buckets is our count per bucket
    // indices is, for each item, the count of the bucket it was placed in, before it was placed there.
    // sublist is the bucket in which a given item was placed.
    unsigned int N = length;
    unsigned int L = NUMBER_OF_PROCESSORS * 2;
    int elemsneeded = ceil((float) N/L);

    for (unsigned int i = 0; i < 10; i++) {
      int range = pivots[i + 1] - pivots[i];
      int j = 0;
      while (buckets[i] >= elemsneeded) {
        pivots[i + 1] += (elemsneeded/buckets[i]) * range;
        elemsneeded = N/L;
        buckets[i] -= elemsneeded;
        j++;
      }
      elemsneeded -= buckets[i];
      pivots[i + 1] += range / 2;
    }

    // /*****************************STEP 2 COMPLETE******************************/
    //
    // // std::cout << "After" << std::endl;
    // // for (unsigned int i = 0; i < 20; i++) {
    // //   std::cout << pivots[i] << std::endl;
    // // }
    // //
    // // int count = 0;
    // // for (int i = 0; i < length; i++) {
    // //   if (in[i] >= pivots[8] && in[i] <= pivots[9]) {
    // //     count++;
    // //   }
    // // }
    // // std::cout << "CPU Count: " << count << std::endl;
    //
    // /****************************STEP 3****************************************/
    //Launch a kernal to count the number of items in each bucket after
    //redefining pivots!

    //Copying things to memory
    //Input/output mallocs
    CHECK(hipMalloc((void**)&d_pivots, pivotsLength * sizeof(unsigned int)));
    CHECK(hipMalloc((void**)&r_buckets, pivotsLength * sizeof(int)));
    CHECK(hipMalloc((void**)&d_in, length * sizeof(unsigned int)));
    CHECK(hipMalloc((void**)&r_indices, length * sizeof(unsigned int)));
    CHECK(hipMalloc((void**)&r_sublist, length * sizeof(unsigned int)));

    //Copying things to memory
    CHECK(hipMemcpy(d_pivots, pivots,
      pivotsLength * sizeof(unsigned int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_in, in,
      length * sizeof(unsigned int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(r_buckets, buckets_count,
      pivotsLength * sizeof(int), hipMemcpyHostToDevice));

    //Launching kernel
    d_count_kernel<<<grid, block>>>(d_pivots, r_buckets, pivotsLength,
      r_indices, r_sublist, d_in, length);

    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(buckets, r_buckets, pivotsLength * sizeof(int), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(indices, r_indices, length * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(sublist, r_sublist, length * sizeof(unsigned int), hipMemcpyDeviceToHost));

    CHECK(hipFree(d_pivots));
    CHECK(hipFree(r_buckets));
    CHECK(hipFree(r_indices));
    CHECK(hipFree(r_sublist));
    CHECK(hipFree(d_in));

    // free(pivots);

    // /***************************STEP 3 COMPLETE********************************/

    // for (int i = 0; i < length; i++) {
    //   std::cout << "item: " << in[i] << " went into bucket " << sublist[i] << " which has pivot " << pivots[sublist[i]] << ". That bucket contains " << buckets[sublist[i]] << " items and this item is at index " << indices[i] << std::endl;
    // }

    //Calculate prefix sums for buckets to find the starting index of each
    //bucket in our final bucketsorted array.
    unsigned int * prefix_buckets = (unsigned int *) Malloc(pivotsLength * sizeof(unsigned int));
    prefix_buckets[0] = buckets[0];
    for (unsigned int i = 1; i < pivotsLength; i++) {
      prefix_buckets[i] = prefix_buckets[i - 1] + buckets[i - 1];
    }

    // for (unsigned int i = 0; i < 30; i++) {
    //   if (i % 10 == 0) {
    //     std::cout << std::endl;
    //   }
    //   std::cout << std::setw(8) << buckets[i] << ", ";
    // }
    // std::cout << std::endl;
    //
    // for (unsigned int i = 0; i < 30; i++) {
    //   if (i % 10 == 0) {
    //     std::cout << std::endl;
    //   }
    //   std::cout << std::setw(8) << prefix_buckets[i] << ", ";
    // }
    // std::cout << std::endl;

    // /***********************STEP 4: BUCKETSORT*********************************/

    CHECK(hipMalloc((void**)&d_in, length * sizeof(unsigned int)));
    unsigned int * d_indices;
    CHECK(hipMalloc((void**)&d_indices, length * sizeof(unsigned int)));
    unsigned int * d_sublist;
    CHECK(hipMalloc((void**)&d_sublist, length * sizeof(unsigned int)));
    unsigned int * r_outputlist;
    CHECK(hipMalloc((void**)&r_outputlist, length * sizeof(unsigned int)));
    unsigned int * d_bucketoffsets;
    CHECK(hipMalloc((void**)&d_bucketoffsets, pivotsLength * sizeof(unsigned int)));

    CHECK(hipMemcpy(d_in, in,
      length * sizeof(unsigned int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_indices, indices, length * sizeof(unsigned int),
            hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_sublist, sublist, length * sizeof(unsigned int),
            hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bucketoffsets, prefix_buckets,
      pivotsLength * sizeof(unsigned int), hipMemcpyHostToDevice));

    d_bucketsort<<<grid, block>>>(d_in, d_indices, d_sublist,
      r_outputlist, d_bucketoffsets, length);

    CHECK(hipDeviceSynchronize());

    unsigned int * outputlist = (unsigned int *) Malloc(length * sizeof(unsigned int));
    CHECK(hipMemcpy(outputlist, r_outputlist, length * sizeof(unsigned int), hipMemcpyDeviceToHost));

    CHECK(hipFree(d_in));
    CHECK(hipFree(d_indices));
    CHECK(hipFree(d_sublist));
    CHECK(hipFree(r_outputlist));
    CHECK(hipFree(d_bucketoffsets));
    free(outputlist);

    free(buckets);
    free(indices);
    free(sublist);
    free(pivots);

    // unsigned long size = 2 * NUMCHARS * sizeof(unsigned char);
    // unsigned long outsize = pow(NUMCHARS, 2) * 3;

    // unsigned char * d_passwords;
    // CHECK(hipMalloc((void**)&d_passwords, size));
    // unsigned char * d_result;
    // CHECK(hipMalloc((void**)&d_result, outsize));


    //Copy the starting passwords array and valid characters to the GPU
    // CHECK(hipMemcpyToSymbol(HIP_SYMBOL(VALID_CHARS), VALID_CHARS_CPU, NUMCHARS * sizeof(char)));
    // CHECK(hipMemcpy(d_passwords, STARTING_PASSES, 2 * NUMCHARS, hipMemcpyHostToDevice));


    // Beginning of Four-way Radix Sort
    // We need a block size of 256
    // dim3 block(BLOCKDIM, 1, 1);
    // dim3 grid(1, 1, 1);

    // d_generate_kernel<<<grid, block>>>(d_passwords, 1, NUMCHARS, d_result);

    // CHECK(hipDeviceSynchronize());
    //
    // unsigned char * passwords = (unsigned char *) Malloc(outsize);
    // CHECK(hipMemcpy(passwords, d_result, outsize, hipMemcpyDeviceToHost));
    //
    // CHECK(hipFree(d_passwords));
    // CHECK(hipFree(d_result));
    //
    // free(passwords);

    //record the ending time and wait for event to complete
    CHECK(hipEventRecord(stop_cpu));
    CHECK(hipEventSynchronize(stop_cpu));
    //calculate the elapsed time between the two events
    CHECK(hipEventElapsedTime(&cpuMsecTime, start_cpu, stop_cpu));
    return cpuMsecTime;
}

__global__ void d_count_kernel(unsigned int * d_pivots,
  int * r_buckets, int pivotsLength, unsigned int * r_indices,
  unsigned int * r_sublist, unsigned int * d_in, int itemCount) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < itemCount) {
      unsigned int element = d_in[idx];
      unsigned int index = pivotsLength/2 - 1;
      unsigned int jump = pivotsLength/4;
      int pivot = d_pivots[index];
      while(jump >= 1) {
        index = (element < pivot) ? (index - jump) : (index + jump);
        pivot = d_pivots[index];
        jump /= 2;
      }
      index = (element < pivot) ? index : index + 1;
      r_sublist[idx] = index;
      r_indices[idx] = atomicAdd(&r_buckets[index], 1);
      // printf("idx: %d, element: %d, r_sublist[idx]: %d, r_indices[idx]: %d, pivot: %d\n", idx, element, r_sublist[idx], r_indices[idx], pivot);
    }
}

__global__ void d_bucketsort(unsigned int * d_in, unsigned int * d_indices,
    unsigned int * d_sublist, unsigned int * r_outputlist,
    unsigned int * d_bucketoffsets, int itemCount) {
      int idx = blockIdx.x * blockDim.x + threadIdx.x;
      if (idx < itemCount) {
        int newpos = d_bucketoffsets[d_sublist[idx]] + d_indices[idx];
        r_outputlist[newpos] = d_in[idx];
      }
}

/*d_generate_kernel
*  Kernal code executed by each thread to generate a list of all possible
*  passwords of length n + 1.  To do this, each thread will work on one element
*  in passwords and append all characters in VALID_CHARS to it. This kernal
*  works in place, so it will alter the input array.
*
*  @params:
*   passwords - array filled with current passwords to build off of.
*   length    - length of the given passwords
*   n         - number of items currently in passwords array
*   d_result  - location to place newly generated passwords.
*/
__global__ void d_sort_kernel() {
  // unsigned long index = blockIdx.x * blockDim.x + threadIdx.x;
}
