#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <openssl/md5.h>
#include "d_hybrid_sort.h"
#include "CHECK.h"
#include "config.h"
#include "wrappers.h"

#define NUMBER_OF_PROCESSORS 1024
#define BLOCK_DIM 256

//prototype for pivot counting kernal
__global__ void d_count_kernel(unsigned int * d_pivots,
  unsigned int * r_buckets, int pivotsLength, unsigned int * r_indices,
  unsigned int * r_sublist, unsigned int * d_in, int itemCount);

//prototype for the kernel
__global__ void d_sort_kernel();

/*d_crack
*
* Sets up and calls the kernal to brute-force a password hash.
*
* @params
*   hash    - the password hash to brute-force
*   hashLen - the length of the hash
*   outpass - the result password to return
*/
float d_sort(unsigned int * in, unsigned int length) {

    hipEvent_t start_cpu, stop_cpu;
    float cpuMsecTime = -1;

    //Use cuda functions to do the timing
    //create event objects
    CHECK(hipEventCreate(&start_cpu));
    CHECK(hipEventCreate(&stop_cpu));
    //record the starting time
    CHECK(hipEventRecord(start_cpu));

    //Find min and max
    unsigned int max = 0;
    unsigned int min = UINT_MAX;
    for (unsigned int i = 0; i < length; i++) {
      if (in[i] < min) {
        min = in[i];
      }
      if (in[i] > max) {
        max = in[i];
      }
    }

    //Compute pivots through linear interpolation
    unsigned int pivotsLength = (NUMBER_OF_PROCESSORS * 2) - 1;
    unsigned int * pivots = new unsigned int[pivotsLength];
    unsigned int * buckets_count = new unsigned int[pivotsLength];
    int slope = (max - min)/pivotsLength;
    for (unsigned int i = 0; i < pivotsLength; i++) {
      pivots[i] = (slope * i);
      buckets_count[i] = 0;
    }

    //Launch a kernal to count the number of items in each bucket so we can
    //refine our pivots later.
    unsigned int * d_pivots;
    CHECK(hipMalloc((void**)&d_pivots, pivotsLength * sizeof(unsigned int)));
    unsigned int * r_buckets;
    CHECK(hipMalloc((void**)&r_buckets, pivotsLength * sizeof(unsigned int)));
    unsigned int * d_in;
    CHECK(hipMalloc((void**)&d_in, length * sizeof(unsigned int)));
    unsigned int * r_indices;
    CHECK(hipMalloc((void**)&r_indices, length * sizeof(unsigned int)));
    unsigned int * r_sublist;
    CHECK(hipMalloc((void**)&r_sublist,
      (pivotsLength + 1) * sizeof(unsigned int)));

    CHECK(hipMemcpy(d_pivots, pivots,
      pivotsLength * sizeof(unsigned int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_in, in,
      length * sizeof(unsigned int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(r_buckets, buckets_count,
      pivotsLength * sizeof(unsigned int), hipMemcpyHostToDevice));

    dim3 block(BLOCKDIM, 1, 1);
    dim3 grid(ceil((float) length/BLOCKDIM), 1, 1);

    d_count_kernel<<<grid, block>>>(d_pivots, r_buckets, pivotsLength,
      r_indices, r_sublist, d_in, length);

    CHECK(hipDeviceSynchronize());

    unsigned int * buckets = (unsigned int *) Malloc(pivotsLength * sizeof(unsigned int));
    CHECK(hipMemcpy(buckets, r_buckets, pivotsLength * sizeof(unsigned int), hipMemcpyDeviceToHost));

    for (int i = 0; i < pivotsLength; i++) {
      if (i % 10 == 0) {
        std::cout << std::endl;
      }
      std::cout << std::setw(8) << buckets[i] << ", ";
    }
    std::cout << std::endl;

    CHECK(hipFree(d_pivots));
    CHECK(hipFree(r_buckets));
    CHECK(hipFree(r_indices));
    CHECK(hipFree(r_sublist));
    CHECK(hipFree(d_in));
    free(buckets);
    free(pivots);

    // unsigned long size = 2 * NUMCHARS * sizeof(unsigned char);
    // unsigned long outsize = pow(NUMCHARS, 2) * 3;

    // unsigned char * d_passwords;
    // CHECK(hipMalloc((void**)&d_passwords, size));
    // unsigned char * d_result;
    // CHECK(hipMalloc((void**)&d_result, outsize));


    //Copy the starting passwords array and valid characters to the GPU
    // CHECK(hipMemcpyToSymbol(HIP_SYMBOL(VALID_CHARS), VALID_CHARS_CPU, NUMCHARS * sizeof(char)));
    // CHECK(hipMemcpy(d_passwords, STARTING_PASSES, 2 * NUMCHARS, hipMemcpyHostToDevice));


    // Beginning of Four-way Radix Sort
    // We need a block size of 256
    // dim3 block(BLOCKDIM, 1, 1);
    // dim3 grid(1, 1, 1);

    // d_generate_kernel<<<grid, block>>>(d_passwords, 1, NUMCHARS, d_result);

    // CHECK(hipDeviceSynchronize());
    //
    // unsigned char * passwords = (unsigned char *) Malloc(outsize);
    // CHECK(hipMemcpy(passwords, d_result, outsize, hipMemcpyDeviceToHost));
    //
    // CHECK(hipFree(d_passwords));
    // CHECK(hipFree(d_result));
    //
    // free(passwords);

    //record the ending time and wait for event to complete
    CHECK(hipEventRecord(stop_cpu));
    CHECK(hipEventSynchronize(stop_cpu));
    //calculate the elapsed time between the two events
    CHECK(hipEventElapsedTime(&cpuMsecTime, start_cpu, stop_cpu));
    return cpuMsecTime;
}

__global__ void d_count_kernel(unsigned int * d_pivots,
  unsigned int * r_buckets, int pivotsLength, unsigned int * r_indices,
  unsigned int * r_sublist, unsigned int * d_in, int itemCount) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < itemCount) {
      unsigned int element = d_in[idx];
      unsigned int index = pivotsLength/2 - 1;
      unsigned int jump = pivotsLength/4;
      int pivot = d_pivots[index];
      while(jump >= 1) {
        index = (element < pivot) ? (index - jump) : (index + jump);
        pivot = d_pivots[index];
        jump /= 2;
      }
      index = (element < pivot) ? index : index + 1;
      r_sublist[idx] = index;
      r_indices[idx] = atomicAdd(&r_buckets[index], 1);
    }
}

/*d_generate_kernel
*  Kernal code executed by each thread to generate a list of all possible
*  passwords of length n + 1.  To do this, each thread will work on one element
*  in passwords and append all characters in VALID_CHARS to it. This kernal
*  works in place, so it will alter the input array.
*
*  @params:
*   passwords - array filled with current passwords to build off of.
*   length    - length of the given passwords
*   n         - number of items currently in passwords array
*   d_result  - location to place newly generated passwords.
*/
__global__ void d_sort_kernel() {
  // unsigned long index = blockIdx.x * blockDim.x + threadIdx.x;
}
